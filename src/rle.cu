#include "hip/hip_runtime.h"
/*
 ============================================================================
 Name        : rle.cu
 Author      : Witaut Bajaryn
 Version     :
 Copyright   : Copyright (c) 2017 Witaut Bajaryn
 Description : Compute sum of reciprocals using STL on CPU and Thrust on GPU
 ============================================================================
 */

#include <iostream>
#include <stdio.h>
#include <unistd.h>
#include <vector>
#include <string>

#include "hemi/hemi.h"
#include "hemi/kernel.h"
#include "hemi/parallel_for.h"

#define HIPCUB_STDERR // For CubDebugExit
#include "cub/util_allocator.cuh"
#include "cub/device/device_scan.cuh"
#include "cub/device/device_run_length_encode.cuh"

using in_elt_t = int;

#define BUILD_NUMBER 14

#define CUB_RLE_MAX_WORKING_INPUT_PIECE_SIZE (274ull * 1024 * 1024)
#define GPU_RLE_MAX_WORKING_INPUT_PIECE_SIZE (237ull * 1024 * 1024)

template<typename elt_t>
struct array
{
	elt_t *data;
	size_t size; // the number of elt_t elements in data

	static array<elt_t> new_on_device(size_t size)
	{
		array<elt_t> d_result{nullptr, size};
		d_result.hipMalloc();
		return d_result;
	}

	static array<elt_t> vector_view_on_host(std::vector<elt_t> &v)
	{
		return array<elt_t>{v.data(), v.size()};
	}

	array<elt_t> subview(size_t offset, size_t subview_size)
	{
		size_t result_size = std::min(subview_size, size - offset);
		return array<elt_t>{data + offset, result_size};
	}

	elt_t &operator[](const size_t i)
	{
		return data[i];
	}

	void hipMalloc()
	{
		checkCuda(::hipMalloc(&data, size * sizeof(*data)));
	}

	void hipFree()
	{
		checkCuda(::hipFree(data));
	}
};

// From https://erkaman.github.io/posts/cuda_rle.html
int cpuRLEImpl(const in_elt_t *in, int n, in_elt_t* symbolsOut, int* countsOut)
{
	if (n == 0)
		return 0; // nothing to compress!

	int outIndex = 0;
	in_elt_t symbol = in[0];
	int count = 1;

	for (int i = 1; i < n; ++i) {
		if (in[i] != symbol) {
			// run is over.
			// So output run.
			symbolsOut[outIndex] = symbol;
			countsOut[outIndex] = count;
			outIndex++;

			// and start new run:
			symbol = in[i];
			count = 1;
		} else {
			++count; // run is not over yet.
		}
	}

	// output last run.
	symbolsOut[outIndex] = symbol;
	countsOut[outIndex] = count;
	outIndex++;

	return outIndex;
}

void cpuRLE(
		array<in_elt_t> in,
		std::vector<in_elt_t> &out_symbols,
		std::vector<int> &out_counts,
		int &out_end)
{
	out_end = cpuRLEImpl(in.data, in.size,
			out_symbols.data(),
			out_counts.data());
}

void inclusive_prefix_sum(array<uint8_t> d_in, array<int> d_out)
{
    hipcub::CachingDeviceAllocator allocator(true);

    void *d_temp_storage = nullptr;
    size_t temp_storage_bytes = 0;
    // Estimate temp_storage_bytes
    CubDebugExit(hipcub::DeviceScan::InclusiveSum(
    		d_temp_storage, temp_storage_bytes,
    		d_in.data, d_out.data, d_in.size,
    		0, true));
    CubDebugExit(allocator.DeviceAllocate(&d_temp_storage, temp_storage_bytes));
    CubDebugExit(hipPeekAtLastError());
    hemi::deviceSynchronize();
    // Run
    std::cout << "Running prefix sum kernel" << std::endl;
    auto err = (hipcub::DeviceScan::InclusiveSum(
    		d_temp_storage, temp_storage_bytes,
    		d_in.data, d_out.data, d_in.size,
    		0, true));
    std::cerr << hipGetErrorString(err) << std::endl;
    checkCuda(err);
    CubDebugExit(err);
    std::cout << "Done" << std::endl;
}

void deviceRLE(
		array<in_elt_t> d_in,
		array<in_elt_t> d_out_symbols,
		array<int> d_out_counts,
		array<int> d_end)
{
	// Idea: https://erkaman.github.io/posts/cuda_rle.html

	auto d_backward_mask = array<uint8_t>::new_on_device(d_in.size);
	hemi::parallel_for(0, d_backward_mask.size, [=] HEMI_LAMBDA(size_t i) {
		if (i == 0) {
			d_backward_mask.data[i] = 1;
			return;
		}
		d_backward_mask.data[i] = d_in.data[i] != d_in.data[i - 1];
	});

	auto d_scanned_backward_mask = array<int>::new_on_device(d_in.size);
	inclusive_prefix_sum(d_backward_mask, d_scanned_backward_mask);

	auto d_compacted_backward_mask = array<int>::new_on_device(d_in.size + 1);
	hemi::parallel_for(0, d_in.size, [=] HEMI_LAMBDA(size_t i) {
		if (i == 0) {
			d_compacted_backward_mask.data[i] = 0;
			return;
		}

		size_t out_pos = d_scanned_backward_mask.data[i] - 1;

		if (i == d_in.size - 1) {
			*d_end.data = out_pos + 1;
			d_compacted_backward_mask.data[out_pos + 1] = i + 1;
		}

		// or if (d_scanned_backward_mask.data[i] !=
		//        d_scanned_backward_mask.data[i - 1])
		if (d_backward_mask.data[i])
			d_compacted_backward_mask.data[out_pos] = i;
	});

	// Not hemi::parallel_for because d_end is only on the device now.
	hemi::launch([=] HEMI_LAMBDA() {
		for (size_t i: hemi::grid_stride_range(0, *d_end.data)) {
			int current = d_compacted_backward_mask.data[i];
			int right = d_compacted_backward_mask.data[i + 1];
			d_out_counts.data[i] = right - current;
			d_out_symbols.data[i] = d_in.data[current];
		}
	});

	// Call it explicitly
	// to make the profiler not show other calls as taking a long time.
	hemi::deviceSynchronize();

	d_compacted_backward_mask.hipFree();
	d_scanned_backward_mask.hipFree();
	d_backward_mask.hipFree();
}

void cubDeviceRLE(
		array<in_elt_t> d_in,
		array<in_elt_t> d_out_symbols,
		array<int> d_out_counts,
		array<int> d_end)
{
	array<uint8_t> d_temp_storage{nullptr, 0};
	// Estimate d_temp_storage.size
	CubDebugExit(hipcub::DeviceRunLengthEncode::Encode(
			d_temp_storage.data, d_temp_storage.size,
			d_in.data,
			d_out_symbols.data, d_out_counts.data, d_end.data, d_in.size));
	d_temp_storage.hipMalloc();
	CubDebugExit(hipcub::DeviceRunLengthEncode::Encode(
			d_temp_storage.data, d_temp_storage.size,
			d_in.data,
			d_out_symbols.data, d_out_counts.data, d_end.data, d_in.size));

	hemi::deviceSynchronize();
}

void gpuRLE(
		array<in_elt_t> in,
		std::vector<in_elt_t> &out_symbols,
		std::vector<int> &out_counts,
		int &out_end,
		bool use_cub_impl = false)
{
	auto d_in = array<in_elt_t>::new_on_device(in.size);
	auto d_out_symbols = array<in_elt_t>::new_on_device(in.size);
	auto d_out_counts = array<int>::new_on_device(in.size);
	auto d_end = array<int>::new_on_device(1);

	checkCuda(hipMemcpy(d_in.data, in.data,
			d_in.size * sizeof(*d_in.data),
			hipMemcpyHostToDevice));

	if (use_cub_impl)
		cubDeviceRLE(d_in, d_out_symbols, d_out_counts, d_end);
	else
		deviceRLE(d_in, d_out_symbols, d_out_counts, d_end);

	checkCuda(hipMemcpy(out_symbols.data(), d_out_symbols.data,
			out_symbols.size() * sizeof(*out_symbols.data()),
			hipMemcpyDeviceToHost));
	checkCuda(hipMemcpy(out_counts.data(), d_out_counts.data,
			out_counts.size() * sizeof(*out_counts.data()),
			hipMemcpyDeviceToHost));
	checkCuda(hipMemcpy(&out_end, d_end.data,
			sizeof(out_end),
			hipMemcpyDeviceToHost));

	d_in.hipFree();
	d_out_symbols.hipFree();
	d_out_counts.hipFree();
	d_end.hipFree();
}

void run_rle_impl(
		array<in_elt_t> in,
		std::vector<in_elt_t> &out_symbols,
		std::vector<int> &out_counts,
		int &out_end,
		bool use_cpu_impl,
		bool use_cub_impl)
{
	if (use_cpu_impl)
		cpuRLE(in, out_symbols, out_counts, out_end);
	else
		gpuRLE(in, out_symbols, out_counts, out_end, use_cub_impl);
}

void append_partial_result(
		std::vector<in_elt_t> &out_symbols,
		std::vector<int> &out_counts,
		std::vector<in_elt_t> &full_out_symbols,
		std::vector<int> &full_out_counts)
{
	size_t offset = 0;

	if (full_out_symbols.size() > 0 && out_symbols.size() > 0) {
		std::cout << "append_partial_result: Piecing together needed"
				  << std::endl;

		size_t prev_full_end{out_symbols.size() - 1};
		if (full_out_symbols[prev_full_end] == out_symbols[0]) {
			full_out_counts[prev_full_end] += out_counts[0];
			offset = 1;
		}
	}

	std::cout << "append_partial_result: Copying partial result" << std::endl;
	std::copy(out_symbols.begin() + offset, out_symbols.end(),
			std::back_inserter(full_out_symbols));
	std::copy(out_counts.begin() + offset, out_counts.end(),
			std::back_inserter(full_out_counts));
}

void rle(
		std::vector<in_elt_t> &in_owner,
		std::vector<in_elt_t> &full_out_symbols,
		std::vector<int> &full_out_counts,
		size_t piece_size,
		bool use_cpu_impl,
		bool use_cub_impl)
{
	array<in_elt_t> full_in = array<in_elt_t>::vector_view_on_host(in_owner);

	for (size_t start = 0; start < in_owner.size(); start += piece_size) {
		array<in_elt_t> in = full_in.subview(start, piece_size);
		std::cout << "Partial in start: " << start
				  << ", size: " << in.size << std::endl;

		// TODO Could actually be allocated once
		std::vector<in_elt_t> out_symbols(in.size);
		std::vector<int> out_counts(in.size);
		int end{0};

		run_rle_impl(in, out_symbols, out_counts, end,
				use_cpu_impl, use_cub_impl);

		out_symbols.resize(end);
		out_counts.resize(end);

		append_partial_result(out_symbols, out_counts,
				full_out_symbols, full_out_counts);
	}
}

bool verify_rle(
		std::vector<in_elt_t> &in,
		std::vector<in_elt_t> &out_symbols,
		std::vector<int> &out_counts)
{
	std::vector<in_elt_t> decompressed{};
	for (size_t i = 0; i < out_symbols.size(); i++)
		for (int j = 0; j < out_counts[i]; j++)
			decompressed.push_back(out_symbols[i]);

	if (decompressed.size() != in.size()) {
		std::cout << "Uncompressed output size (" << decompressed.size()
				  << ") != input size (" << in.size() << ")." << std::endl;
		return false;
	}

	for (size_t i = 0; i < decompressed.size(); i++)
		if (decompressed[i] != in[i]) {
			std::cout << "Uncompressed output element[" << i
					  << "] (" << decompressed[i]
					  << ") != input element (" << in[i] << ")." << std::endl;
			return false;
		}
	return true;
}

std::vector<in_elt_t> generate_input(size_t size)
{
	std::vector<in_elt_t> result{};

	int run_count = 1;
	int run_value = 0;
	int run_i = 0;
	for (size_t i = 0; i < size; i++) {
		result.push_back(run_value);

		run_i++;

		if (run_i >= run_count) {
			run_count++;
			run_value++;
			run_i = 0;
		}
	}

	return result;
}

void parse_args(
		int argc,
		char *argv[],
		size_t *input_size,
		size_t *input_piece_size,
		bool *use_cpu_impl,
		bool *use_cub_impl)
{
	int opt;

	while ((opt = getopt(argc, argv, "cus:p:")) != -1) {
		switch (opt) {
		case 'c':
			*use_cpu_impl = true;
			break;
		case 'u':
			*use_cub_impl = true;
			*use_cpu_impl = false;
			break;
		case 's':
			*input_size = atoll(optarg);
			break;
		case 'p':
			*input_piece_size = atoll(optarg);
			break;
		default:
			fprintf(stderr, "Usage: %s [-c|-u] [-s input_size] [-p input_piece_size]\n", argv[0]);
			exit(EXIT_FAILURE);
		}
	}
}

void assert_safe_piece_size(
		size_t piece_size,
		size_t max_safe_piece_size,
		std::string impl_name)
{
	if (piece_size > max_safe_piece_size) {
		std::cout << "Compressing the input by pieces of more than "
				  << max_safe_piece_size << " elements requested. "
				  << impl_name << " produces incorrect results"
				  << " or runs out of memory for such inputs."
				  << " Terminating without running anything." << std::endl;
		exit(EFBIG);
	}
}

int main(int argc, char *argv[])
{
	size_t input_size = 200llu * 1024 * 1024;
	size_t input_piece_size = 200llu * 1024 * 1024;
	bool use_cpu_impl = false;
	bool use_cub_impl = false;

	parse_args(argc, argv,
			&input_size, &input_piece_size,
			&use_cpu_impl, &use_cub_impl);

	std::cout << "Build " << BUILD_NUMBER << std::endl;
	std::cout << "Generating an input with " << input_size
			  << " elements (" << input_size * sizeof(in_elt_t) << " bytes)"
			  << std::endl;
	std::cout << "Will compress " << input_piece_size << " elements ("
			  << input_piece_size * sizeof(in_elt_t) << " bytes) at a time"
			  << std::endl;

	if (use_cpu_impl)
		std::cout << "Using the CPU implementation" << std::endl;
	else if (use_cub_impl)
		std::cout << "Using the Cub GPU implementation" << std::endl;
	else
		std::cout << "Using the GPU implementation" << std::endl;

	if (use_cub_impl)
		assert_safe_piece_size(
				input_piece_size, CUB_RLE_MAX_WORKING_INPUT_PIECE_SIZE,
				"CUB RLE");
	else if (!use_cpu_impl)
		assert_safe_piece_size(
				input_piece_size, GPU_RLE_MAX_WORKING_INPUT_PIECE_SIZE,
				"This GPU RLE implementation");

	std::vector<in_elt_t> in_owner = generate_input(input_size);

	std::vector<in_elt_t> out_symbols{};
	//out_symbols.reserve(in_owner.size());
	std::vector<int> out_counts{};
	//out_counts.reserve(in_owner.size());

	rle(in_owner, out_symbols, out_counts,
			input_piece_size,
			use_cpu_impl, use_cub_impl);

	/*
	std::cout << "[";
	for (int i = 0; i < out_symbols.size(); i++)
		std::cout << "(" << out_counts[i]
			 << ", " << out_symbols[i]
			 << "), ";
	std::cout << "]" << std::endl;
	*/

	std::cout << "Verifying the output" << std::endl;
	if (verify_rle(in_owner, out_symbols, out_counts))
		std::cout << "The output is correct." << std::endl;
	else
		std::cout << "The output is INCORRECT." << std::endl;

	return 0;
}
